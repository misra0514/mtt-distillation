#include "hip/hip_runtime.h"
#include <torch/extension.h>
// #include <torch/extension.h>
#include <hip/hip_runtime.h>
// #include <iostream>
#include <stdint.h>
// #include "kernel.h"
// #include "hip/hip_runtime.h"
#include "unistd.h"
#include "iostream"
#include <cstdint>
#include <ATen/cuda/HIPContext.h>

using namespace::std;

void launch_add2(float hitRatio,  uintptr_t stream_handle, at::Tensor base) {
// hipStream_t hip_stream = reinterpret_cast<hipStream_t>(stream_handle);
hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

int l2_cache_size;
hipDeviceGetAttribute(&l2_cache_size, hipDeviceAttributeL2CacheSize, 0);
int num_bytes = 0;
num_bytes = min((int)(1.5 * base.numel() * base.element_size()), l2_cache_size);
hitRatio = max((float)1.0, (float)(num_bytes/ l2_cache_size));
cout<<"+++++++++++"<<endl<<num_bytes<<"  "<<l2_cache_size<<"  "<<hitRatio<<"  "<<endl;

hipLaunchAttributeValue stream_attribute;                                         // Stream level attributes data structure
stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(base.data_ptr()); // Global Memory data pointer
stream_attribute.accessPolicyWindow.num_bytes = 512;                    // Number of bytes for persisting accesses.
//                                                                               // (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
stream_attribute.accessPolicyWindow.hitRatio  = 1.0;                          // Hint for L2 cache hit ratio for persisting accesses in the num_bytes region
stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting; // Type of access property on cache hit
stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyPersisting;  // Type of access property on cache miss.
hipError_t err = hipStreamSetAttribute(0, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);    

std::cout << "Error setting stream attribute: " << hipGetErrorString(err) << std::endl;
}